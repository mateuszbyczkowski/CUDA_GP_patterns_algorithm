#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>

#include "thrust/host_vector.h"
#include "thrust/device_vector.h"
#include "thrust/scan.h"
#include "thrust/scatter.h"
#include "thrust/sort.h"
#include "thrust/functional.h"
#include "thrust/reduce.h"
#include "thrust/copy.h"
#include "thrust/remove.h"
#include "thrust/device_ptr.h"
#include <thrust/execution_policy.h>

#include <math.h>
#include "device_atomic_functions.h"
#include <set>

using namespace std;

__host__ __device__ long getV(int up, int down) {
	long result = 1;
	for (int i = up - down + 1; i <= up; i++) {
		result *= i;
	}
	return result;
}

__global__ void func(int* output, int* sequence, int* pattern, int* uniqueSequence,
					int numberOfDifferentInSequence, int numberOfDifferentInPattern, int patternSize, int sequenceSize) {
	//prepare variables
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int localTid = tid;

	int* localUniqueSequence = new int[numberOfDifferentInSequence]();
	localUniqueSequence = uniqueSequence;

	//count variant for thread
	int* variant = new int[numberOfDifferentInPattern]();

	for (int x = 0; x < patternSize; x++) {
		int v = getV(numberOfDifferentInSequence - x - 1, numberOfDifferentInPattern - x - 1);
		int t = tid / v;
		variant[x] = localUniqueSequence[t];

		for (int i = t; i < numberOfDifferentInSequence; i++) {
			localUniqueSequence[i] = localUniqueSequence[i + 1];
		}

		localUniqueSequence[numberOfDifferentInSequence - 1] = 0;

		tid = tid % v;
	}

	int* finalPattern = new int[patternSize]();

	//subsitution
	for (int i = 0; i < patternSize; i++) {
		finalPattern[i] = variant[pattern[i] - 1];
	}
	
	//find pattern in sequence
	int counter = 0;
	int counter2 = 0;
	for (int i = 0; i < patternSize; i++) {
		for (int k = counter2; k < sequenceSize; k++) {
			if (counter <= i) {
				if (finalPattern[i] == sequence[k]) {
					counter2 = k + 1;
					counter++;
				}
			}
		}
	}

	/*localSequence = sequence;
	int localSequenceSize = sequenceSize;
	int counter = 0;
	int nextII = 0;
	for (int i = 0; i < patternSize; i++) {
		for (int ii = nextII; ii < localSequenceSize; ii++) {
			if (counter <= i) {
				if (finalPattern[i] == localSequence[ii]) {
					nextII = ii + 1;
					counter++;
				}
			}
		}
	}*/

	//pattern found
	if (counter == patternSize) {
		for (int i = 0; i < patternSize; i++) {
			output[localTid * patternSize + i] = finalPattern[i];
		}
	}

	__syncthreads();
}

int countUniqueLetters(thrust::host_vector<int> sequence) {
	set<int> uniqueLettersInSequence;

	for (int i = 0; i < sequence.size(); i++) {
		uniqueLettersInSequence.insert(sequence[i]);
	}

	return uniqueLettersInSequence.size();
}

thrust::host_vector<int> getUniqueSequence(thrust::host_vector<int> sequence) {
	set<int> uniqueInSequence;

	for (int i = 0; i < sequence.size(); i++) {
		uniqueInSequence.insert(sequence[i]);
	}
	thrust::host_vector<int> uniqueInSequenceVector(uniqueInSequence.begin(), uniqueInSequence.end());
	return uniqueInSequenceVector;
}

void projekt() {
	thrust::host_vector<int> h_tab_out;
	thrust::device_vector<int> d_tab_out;

	thrust::device_vector<int> sequence_device;
	thrust::device_vector<int> pattern_device;
	thrust::device_vector<int> uniqueSequence_device;

	thrust::host_vector<int> pattern;
	pattern.push_back(1);
	pattern.push_back(2);
	pattern.push_back(2);
	pattern.push_back(1);
	
	thrust::host_vector<int> sequence;
	sequence.push_back(1);
	sequence.push_back(2);
	sequence.push_back(4);
	sequence.push_back(3);
	sequence.push_back(5);
	sequence.push_back(3);
	sequence.push_back(6);
	sequence.push_back(2);
	sequence.push_back(1);

	int numberOfDifferentInSequence = countUniqueLetters(sequence);
	int numberOfDifferentInPattern = countUniqueLetters(pattern);

	int sequenceSize = sequence.size();
	int patternSize = pattern.size();

	int variantsNumber = getV(numberOfDifferentInSequence, numberOfDifferentInPattern);

	//---------------------------------------------------------------------------------------------------------------//
	cout << "Liczba roznych w patternie: " << numberOfDifferentInPattern << endl;
	cout << "Liczba roznych w sekwencji: " << numberOfDifferentInSequence << endl;
	cout << "Liczba wariancji: " << variantsNumber << endl;
	//---------------------------------------------------------------------------------------------------------------//

	thrust::host_vector<int> uniqueSequence = getUniqueSequence(sequence);

	dim3 dimBlock(variantsNumber);
	dim3 dimGrid(1);

	h_tab_out.resize(variantsNumber * patternSize);
	d_tab_out.resize(variantsNumber * patternSize);

	uniqueSequence_device.resize(uniqueSequence.size());
	sequence_device.resize(sequence.size());
	pattern_device.resize(pattern.size());

	uniqueSequence_device = uniqueSequence;
	sequence_device = sequence;
	pattern_device = pattern;

	func << <dimGrid, dimBlock >> > (
		d_tab_out.data().get(),
		sequence_device.data().get(),
		pattern_device.data().get(),
		uniqueSequence_device.data().get(),
		numberOfDifferentInSequence,
		numberOfDifferentInPattern,
		patternSize,
		sequenceSize
		);

	h_tab_out = d_tab_out; //Kopiowanie device->host

	for (int s = 0; s < h_tab_out.size(); s++) {
		if (s % patternSize == 0) {
			cout << endl;
		}
		cout << h_tab_out[s];
	}
}

int main() {
	projekt();
	return EXIT_SUCCESS;
}